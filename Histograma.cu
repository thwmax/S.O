#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define HIST_LENGHT 256
#define NUMBER_OF_THREADS 512


int main()
{
	/**Guarda hora de inicio**/
	struct timeval before , after;
	gettimeofday(&before , NULL);
	
	int matrix_dim, array_lenght, *data_h, hist_h[HIST_LENGHT];
	int i;

	/** Ficheros de entrada y salida **/
	FILE *in_f = fopen("entrada", "r");
	FILE *out_f = fopen("salida", "w");
	
	/** Leer el primer numero que determina el tamano de la matriz **/
	fscanf(in_f, "%d", &matrix_dim);
	array_lenght = matrix_dim * matrix_dim;

	data_h = (int *)malloc(array_lenght * sizeof(int));
    for (i = 0; i < array_lenght && fscanf(in_f, "%d", &data_h[i]) == 1; ++i);

    CUDA_Hist(data_h, hist_h, array_lenght);

	return 0;
}

void CUDA_Hist(int *data_h, int *hist_h, int array_lenght)
{
	int *data_d, *hist_d, blocks;
	int block_size = NUMBER_OF_THREADS;

	hipMalloc((void **) &data_d, array_lenght * sizeof(int));
    hipMalloc((void **) &hist_d, HIST_LENGHT * sizeof(int));

    hipMemcpy(data_d, data_h, array_lenght * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(hist_d, 0, HIST_LENGHT * sizeof(int));

    blocks = ceil((float)array_lenght/block_size);

    GPUfuncion <<<blocks, block_size>>> (hist_d, data_d, array_lenght);

    hipMemcpy(hist_h, hist_d, HIST_LENGHT * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(hist_d);
    return;
}

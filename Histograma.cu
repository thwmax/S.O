#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define HIST_LENGTH 256
#define NUMBER_OF_THREADS 512

/** Maneja la memoria del dispositivo y el huesped, ademas
 ** de la transferencia de datos entre estos dos **/
void CUDA_Hist(int *data_h, int *hist_h, int array_length);

/** Kernel CUDA, realiza la suma resultante en un histograma **/
__global__ void GPUfuncion(int *hist, int *data, int max)
{
	int t = threadIdx.x;
	int b = blockIdx.x;
	int B = blockDim.x;
	int buffer;

	/** Acceso a memoria compartida es 100x mas rapida que
	 ** memoria global, por lo tanto se crea un arreglo local al
	 ** bloque **/
	__shared__ int hist_temp[HIST_LENGTH];
	if (t < HIST_LENGTH)
	{
		hist_temp[t] = 0;
	}
	/** Barrera para asegurar integridad del arreglo compartido **/
	__syncthreads();

	int index = b * B + t;
	
	if (index < max)
	{
		/** Rescata el valor de la memoria global para cada hebra
		 ** de ejecucion **/
		buffer = data[index];
		atomicAdd(&(hist_temp[buffer]), 1);
		__syncthreads();
		/** Traspasa el resultado a la memoria global **/
		if (t < HIST_LENGTH)
			atomicAdd(&(hist[t]), hist_temp[t]);
	}
	else
		return;
}

int main(int argc, char *argv[])
{	
	float elapsedTime;
	hipEvent_t start, stop;

	/** Comienza a registrar el tiempo de ejecucion **/
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int matrix_dim, array_length, *data_h, hist_h[HIST_LENGTH];
	int i;

	/** Ficheros de entrada y salida **/
	FILE *in_f = fopen(argv[1], "r");
	FILE *out_f = fopen("salida", "w");
	
	/** Leer el primer numero que determina el tamano de la matriz **/
	fscanf(in_f, "%d", &matrix_dim);
	array_length = matrix_dim * matrix_dim;

	/** Se declara arreglo dinamico que contiene a la matriz serializada **/
	data_h = (int *)malloc(array_length * sizeof(int));
	for (i = 0; i < array_length && fscanf(in_f, "%d", &data_h[i]) == 1; ++i);

	CUDA_Hist(data_h, hist_h, array_length);

	/** Imprime en la salida estandar el histograma resultante **/
	for (i = 0; i < 256; i++)
	{
		if (i == 255)
			fprintf(out_f, "%d", hist_h[i]);
		else
			fprintf(out_f, "%d\n", hist_h[i]);
	}
	
	fclose(in_f);
	fclose(out_f);
	
	/** Mediante CUDA API events se calcula el tiempo de ejecucion **/
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Tiempo de ejecucion: %f [ms]\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);	
	return 0;
}

void CUDA_Hist(int *data_h, int *hist_h, int array_length)
{
	int *data_d, *hist_d, blocks;
	int block_size = NUMBER_OF_THREADS;

	hipMalloc((void **) &data_d, array_length * sizeof(int));
	hipMalloc((void **) &hist_d, HIST_LENGTH * sizeof(int));

	hipMemcpy(data_d, data_h, array_length * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(hist_d, 0, HIST_LENGTH * sizeof(int));

	blocks = ceil((float)array_length/block_size);

	GPUfuncion <<<blocks, block_size>>> (hist_d, data_d, array_length);

	hipMemcpy(hist_h, hist_d, HIST_LENGTH * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(data_d);
	hipFree(hist_d);
	return;
}
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define HIST_LENGTH 256
#define NUMBER_OF_THREADS 512

void CUDA_Hist(int *data_h, int *hist_h, int array_length);

__global__ void GPUfuncion(int *hist, int *data, int max)
{
	int t = threadIdx.x;
	int b = blockIdx.x;
	int B = blockDim.x;
	int buffer;

	__shared__ int hist_temp[HIST_LENGTH];
	if (t < HIST_LENGTH)
	{
		hist_temp[t] = 0;
	}
	__syncthreads();

	int index = b * B + t;
	
	if (index < max)
	{
		buffer = data[index];
		atomicAdd(&(hist_temp[buffer]), 1);
		__syncthreads();
		if (t < HIST_LENGTH)
			atomicAdd(&(hist[t]), hist_temp[t]);
	}
	else
		return;
}

int main(int argc, char *argv[])
{	
	float elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int matrix_dim, array_length, *data_h, hist_h[HIST_LENGTH];
	int i;

	/** Ficheros de entrada y salida **/
	FILE *in_f = fopen(argv[1], "r");
	FILE *out_f = fopen("salida", "w");
	
	/** Leer el primer numero que determina el tamano de la matriz **/
	fscanf(in_f, "%d", &matrix_dim);
	array_length = matrix_dim * matrix_dim;

	data_h = (int *)malloc(array_length * sizeof(int));
	for (i = 0; i < array_length && fscanf(in_f, "%d", &data_h[i]) == 1; ++i);

	CUDA_Hist(data_h, hist_h, array_length);

	for (i = 0; i < 256; i++)
	{
		if (i == 255)
			fprintf(out_f, "%d", hist_h[i]);
		else
			fprintf(out_f, "%d\n", hist_h[i]);
	}
	
	fclose(in_f);
	fclose(out_f);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Tiempo de ejecucion: %f [ms]\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);	
	return 0;
}

void CUDA_Hist(int *data_h, int *hist_h, int array_length)
{
	int *data_d, *hist_d, blocks;
	int block_size = NUMBER_OF_THREADS;

	hipMalloc((void **) &data_d, array_length * sizeof(int));
	hipMalloc((void **) &hist_d, HIST_LENGTH * sizeof(int));

	hipMemcpy(data_d, data_h, array_length * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(hist_d, 0, HIST_LENGTH * sizeof(int));

	blocks = ceil((float)array_length/block_size);

	GPUfuncion <<<blocks, block_size>>> (hist_d, data_d, array_length);

	hipMemcpy(hist_h, hist_d, HIST_LENGTH * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(data_d);
	hipFree(hist_d);
	return;
}
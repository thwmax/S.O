#include <stdio.h>
#include <hip/hip_runtime.h>

#define HIST_LENGTH 256
#define THREADS_P_BLOCK 512

int main(int argc, char *argv[])
{
	float elapsed_Time;
	int i, data_length, *data_h;
	unsigned int hist_h[HIST_LENGTH];
	int *data_d, *hist_d, blocks;
	hipEvent_t start, stop;

	/** Timers **/
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	/** Input de datos **/
	FILE *in_f = fopen(argv[1], "r");
	fscanf(in_f, "%d", &data_length);
	data_length *= data_length;

	/** Declaracion dinamica del tamano del arreglo dependiendo de la matriz **/
	data_h = (int *)malloc(data_length * sizeof(int));
	for (i = 0; i < data_length && fscanf(in_f, "%d", &data_h[i]) == 1; ++i);
	fclose(in_f);

	/** Alloc para la memoria en GPU **/
	hipMalloc((void **) &data_d, data_length * sizeof(int));
	hipMemcpy(data_d, data_h, data_length * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **) &hist_d, HIST_LENGTH * sizeof(int));
	hipMemset(hist_d, 0, HIST_LENGTH * sizeof(int));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	blocks = prop.multiProcessorCount;
	//histogram_kernel<<blocks*2, 256>>(data_d, data_length, hist_d);

	//cudaMemcpy(hist_h, hist_d, 256 * sizeof(int), cudaMemcpyDeviceToHost);

	hipFree(data_d);
	hipFree(hist_d);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_Time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Tiempo de ejecucion: %3.3f ms\n", elapsed_Time);
	return 0;
}

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define HIST_LENGTH 256
#define NUMBER_OF_THREADS 512

void CUDA_Hist(int *data_h, int *hist_h, int array_length);

__global__ void GPUfuncion(int *hist, int *data, int max)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int t = threadIdx.x;

	__shared__ int local_hist[256];

	if (t < 256)
		local_hist[t] = 0;

	__syncthreads();

	if (i < max) {
		int aux = data[i];
		atomicAdd(&local_hist[aux], 1);
	}

	__syncthreads();

	if (t < 256)
		atomicAdd(&hist[t], local_hist[t]);
}

int main(int argc, char *argv[])
{	
	float elapsedTime;
	hipEvent_t start, stop;

	int matrix_dim, array_length, *data_h, hist_h[HIST_LENGTH];
	int i;

	/** Ficheros de entrada y salida **/
	FILE *in_f = fopen(argv[1], "r");
	FILE *out_f = fopen("salida", "w");
	
	/** Leer el primer numero que determina el tamano de la matriz **/
	fscanf(in_f, "%d", &matrix_dim);
	array_length = matrix_dim * matrix_dim;

	data_h = (int *)malloc(array_length * sizeof(int));
	for (i = 0; i < array_length && fscanf(in_f, "%d", &data_h[i]) == 1; ++i);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	CUDA_Hist(data_h, hist_h, array_length);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);	

	for (i = 0; i < 256; i++)
	{
		if (i == 255)
			fprintf(out_f, "%d", hist_h[i]);
		else
			fprintf(out_f, "%d\n", hist_h[i]);
	}
	
	fclose(in_f);
	fclose(out_f);
	
	printf("Tiempo de ejecucion: %f [ms]\n", elapsedTime);

	return 0;
}

void CUDA_Hist(int *data_h, int *hist_h, int array_length)
{
	int *data_d, *hist_d, blocks;
	int block_size = NUMBER_OF_THREADS;

	hipMalloc((void **) &data_d, array_length * sizeof(int));
	hipMalloc((void **) &hist_d, HIST_LENGTH * sizeof(int));

	hipMemcpy(data_d, data_h, array_length * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(hist_d, 0, HIST_LENGTH * sizeof(int));
	
	hipMemcpy(hist_d, hist_h, HIST_LENGTH * sizeof(int), hipMemcpyHostToDevice);

	blocks = (int)ceil(array_length/512.0);

	GPUfuncion <<<blocks, block_size, 256 * sizeof(int)>>> (hist_d, data_d, array_length);

	hipFree(data_d);
	hipFree(hist_d);
	return;
}
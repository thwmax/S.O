#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define PI 3.1415

void gauss (double sigma, double gauss_matrix[][5]);
void gpuComputing(double gauss_matrix[][5], int** image_matrix, int** final_matrix, int height, int width);

__global__ void kernel(int* image, int* final, double* gauss, int pitch, int pitch_i, int pitch_f, int height, int width)
{
	int i, j, abs_Pos, c, r;
	int x, y, aux, aux2;
	double gauss_element;
	int image_row, image_element;
	int tid = threadIdx.x;

	float result;

	__shared__ int fila;

	fila = blockIdx.x + 2;

	while(tid < width - 4)
	{
		result = 0;
		abs_Pos = fila * width + (tid +2);
		x = abs_Pos % width;

		for (r = 0; r < 5; ++r) {
			aux = r - 2;
        	double* gauss_row = (double*)((char*)gauss + r * pitch);
        	int* image_row = (int*)((char*)image + ((fila + aux) * pitch));
        	for (c = 0; c < 5; ++c)
        	{
        		aux2 = c - 2;
            	gauss_element = gauss_row[c];
            	image_element = image_row[x + aux2];
            	result += (gauss_element * (double)image_element)/273.0;
				
        	}
		}
		
		int* final_row = (int*)((char*)final +  blockIdx.x * pitch_f);
        final_row[tid] = (int)result;
        //printf("\nResultado %d, thread: %d, Bloque: %d\n", final_row[tid], tid, blockIdx.x);
        tid += 512;
    }	
    return;
}


int main(int argc, char *argv[])
{
	int width, height, i, j;
	int **image_matrix, **final_matrix;
	int **auxiliar_matrix;
	double gauss_matrix[5][5];
	double sigma = strtod(argv[2], NULL);

	int *temp, *temp2, *temp3;

	FILE *in_f = fopen(argv[1], "r");
	
	/** Leer el primer numero que determina el tamano de la matriz **/
	fscanf(in_f, "%d", &width);
	fscanf(in_f, "%d", &height);

	image_matrix = (int**)malloc(width * sizeof(int*));
	temp = (int*)malloc(width * height * sizeof(int));
	for(i = 0; i < width; i++)
		image_matrix[i] = temp + (i * height);

	auxiliar_matrix = (int**)malloc((width + 4) * sizeof(int*));
	temp2 = (int*)malloc((width + 4) * (height + 4) * sizeof(int));
	for(i = 0; i < (width + 4); i++)
		auxiliar_matrix[i] = temp2 + (i * (height + 4));

	final_matrix = (int**)malloc(width * sizeof(int*));
	temp3 = (int*)malloc(width * height * sizeof(int));
	for(i = 0; i < width; i++)
		final_matrix[i] = temp3 + (i * height );

	for (i = 0; i < width; i++)
	{
		for(j = 0; j < height; j++)
			fscanf(in_f, "%d", &image_matrix[i][j]);
	}
	fclose(in_f);
	gauss(sigma, gauss_matrix);
	
	/** Mover la matriz **/
	for(i = 2; i < width + 2; i++)
	{
		for(j = 2; j < height + 2; j++){
			auxiliar_matrix[i][j] = image_matrix[i-2][j-2];
		}
	}

	/** Copiando filas **/
	for(i = 2; i < width + 2; i++)
	{
		auxiliar_matrix[i][0] = auxiliar_matrix[i][4];
		auxiliar_matrix[i][1] = auxiliar_matrix[i][3];
		auxiliar_matrix[i][height + 2] = auxiliar_matrix[i][height];
		auxiliar_matrix[i][height + 3] = auxiliar_matrix[i][height-1];
	}

	/** Copiando columnas **/
	for(i = 0; i < height + 4; i++)
	{
		auxiliar_matrix[0][i] = auxiliar_matrix[4][i];
		auxiliar_matrix[1][i] = auxiliar_matrix[3][i];
		auxiliar_matrix[width + 2][i] = auxiliar_matrix[width][i];
		auxiliar_matrix[width + 3][i] = auxiliar_matrix[width-1][i];
	}

	gpuComputing(gauss_matrix, auxiliar_matrix, final_matrix, height, width);
    
	FILE *out = fopen("salida", "w");
	for (i = 0; i < height; i++)
	{
        for (j = 0; j < width; j++)
        {
			if ( j != width - 1)
				fprintf(out, "%d\t", final_matrix[i][j]);
			else
				fprintf(out, "%d\n", final_matrix[i][j]);
        }
	}

	fclose(out);
	free(temp);
	free(temp2);
	free(temp3);

	return 0;
}

void gauss(double sigma, double gauss_matrix[][5])
{
	int i, j;
	double x = -2.0, y = 2.0;
	double u, v, varianza;

	for(i = 0; i < 5; i++)
	{
		for(j = 0; j < 5; j++)
		{
			u = pow(x,2);
			v = pow(y,2);
			varianza = pow(sigma,2);

			gauss_matrix[i][j] = exp((-u-v)/(2*varianza))/(2*PI*varianza);
			x++;
		}
		y--;
		x = -2.0;
	}

	for(i = 0; i < 5; i++)
		for(j = 0; j < 5; j++)
			gauss_matrix[i][j] = ceil((gauss_matrix[i][j]*273.0));
	return;
}


void gpuComputing(double gauss_matrix[][5], int** image_matrix, int** final_matrix, int height, int width)
{
	int *d_image, *d_final;
	double *d_gauss;
	int blocks, threads;
	int dimension = height * width;

	size_t pitch, pitch_i, pitch_f;

	hipMallocPitch(&d_gauss, &pitch, 5 * sizeof(double), 5);
	hipMallocPitch(&d_image, &pitch_i, (width + 4) * sizeof(int), (height + 4));
	hipMallocPitch(&d_final, &pitch_f, width * sizeof(int), height);
	
	hipMemcpy2D(d_gauss, pitch, *gauss_matrix, 5 * sizeof(double), 5 * sizeof(double), 5, hipMemcpyHostToDevice);
	hipMemcpy2D(d_image, pitch_i, *image_matrix, (width + 4) * sizeof(int), (width + 4) * sizeof(int), (height + 4), hipMemcpyHostToDevice);

	kernel<<<height, 512>>>(d_image, d_final, d_gauss, pitch, pitch_i, pitch_f, (height + 4), (width + 4));
	hipMemcpy2D(*final_matrix, width*sizeof(int), d_final, pitch_f, width*sizeof(int), height, hipMemcpyDeviceToHost);
	
	hipFree(d_final);
	hipFree(d_image);
	hipFree(d_gauss);

	return;
}
